#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "EXECUTE_HELPER.h"

__device__ int getGlobalIdx_1D_2D(){
    return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void cuda_Hello_World(){
    int tid = getGlobalIdx_1D_2D();
    printf("Hello World from thread ID:%d\n",tid);
}
//op_nop - no operator
//op_mat_inv -  matrix inversion
__device__ void op_mat_inv(){

}
//op_mat_mul - matrix multiplication
__device__ void op_mat_mul(){

}
//op mat add - matrix addition
__device__ void op_mat_add(){

}
//op_vec_mul - multiply two vectors
__device__ void op_vec_mul(){

}
//op_vec_add - add two vectors
__global__ void op_vec_add(double * result,double * vector0,double * vector1,double scalar,int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size){
        result[tid] = vector0[tid] + scalar * vector1[tid];
    }
}

void cuda_kernel_vector_addition(double * result,double * vector0,double * vector1, double scalar,int size){
    
    double * device_vector0;
    double * device_vector1;
    double * device_result;

    hipMalloc((void**)&device_vector0,size * sizeof(double) );
    hipMalloc((void**)&device_vector1,size * sizeof(double) );
    hipMalloc((void**)&device_result,size * sizeof(double));
    //copy the input vectors into the device
    hipMemcpy(device_vector0,vector0,size * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_vector1,vector1,size * sizeof(double),hipMemcpyHostToDevice);

    op_vec_add<<<1,size>>>(device_result,device_vector0,device_vector1,scalar,size);

    hipDeviceSynchronize();

    hipMemcpy(result,device_result,size * sizeof(double),hipMemcpyDeviceToHost);

    hipFree(device_vector0);
    hipFree(device_vector0);
    hipFree(device_result);
}



void Execute_Helper_Kernel(int){
    //create cublas Handle here
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return;
    }else{
        printf("CUBLAS initialized succesfully");
    }
    //create and call the schedhuler here

    //return the result back into the parameter
    hipDeviceSynchronize();
} 
