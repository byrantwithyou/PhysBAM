#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <cstring>

__device__ int getGlobalIdx_1D_2D(){
    return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void cuda_Hello_World(){
    int tid = getGlobalIdx_1D_2D();
    printf("Hello World from thread ID:%d\n",tid);
}

void Hello_World(int){
    dim3 block(3,3);
    dim3 grid(1);
    cuda_Hello_World<<<grid,block>>>();
    hipDeviceSynchronize();
} 
