#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "EXECUTE_HELPER.h"

__device__ int getGlobalIdx_1D_2D(){
    return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void cuda_Hello_World(){
    int tid = getGlobalIdx_1D_2D();
    printf("Hello World from thread ID:%d\n",tid);
}
//op_vec_add - add two vectors
__global__ void op_vec_add(double * result,double * vector0,double * vector1,double scalar,int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size){
        result[tid] = vector0[tid] + scalar * vector1[tid];
    }
}

void cuda_kernel_vector_addition(double * result,double * vector0,double * vector1, double scalar,int size){
    
    double * device_vector0;
    double * device_vector1;
    double * device_result;

    hipMalloc((void**)&device_vector0,size * sizeof(double) );
    hipMalloc((void**)&device_vector1,size * sizeof(double) );
    hipMalloc((void**)&device_result,size * sizeof(double));
    //copy the input vectors into the device
    hipMemcpy(device_vector0,vector0,size * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_vector1,vector1,size * sizeof(double),hipMemcpyHostToDevice);

    op_vec_add<<<1,size>>>(device_result,device_vector0,device_vector1,scalar,size);

    hipDeviceSynchronize();

    hipMemcpy(result,device_result,size * sizeof(double),hipMemcpyDeviceToHost);

    hipFree(device_vector0);
    hipFree(device_vector0);
    hipFree(device_result);
}

void cuda_dgemm_helper(bool at,bool bt,int m, int n, int k,const double * alpha,const double * A,int lda,const double * B,int ldb,const double * beta,double * C,double ldc){
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    status  = hipblasDgemm(handle,at?HIPBLAS_OP_T:HIPBLAS_OP_N,bt?HIPBLAS_OP_T:HIPBLAS_OP_N,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("TIMES MM not working");
    }

    hipblasDestroy(handle);

    return;
}

void cuda_dgemm(bool at,bool bt,int m, int n, int k,const double * alpha,const double * A,int lda,const double * B,int ldb,const double * beta,double * C,double ldc){
    //allocate device variables here
    double * device_A;
    double * device_B;
    double * device_C;

    hipMalloc(&device_A,m*k*sizeof(double));
    hipMalloc(&device_B,n*k*sizeof(double));
    hipMalloc(&device_C,m*n*sizeof(double));

    hipMemcpy(device_A,A,m * k * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_B,B,n* k * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_C,C,m * n * sizeof(double),hipMemcpyHostToDevice);

    cuda_dgemm_helper(at,bt,m,n,k,alpha,device_A,lda,device_B,ldb,beta,device_C,ldc);
    
    hipMemcpy(C,device_C,m * n * sizeof(double),hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    return;
}

void cuda_gemv_helper(bool t,int m, int n,const double * alpha,const double * A, int lda, const double * x, int cx, const double * beta,double * y,int incy)
{
    hipblasStatus_t status;
    hipblasHandle_t handle;
    
    status = hipblasCreate(&handle);
    status = hipblasDgemv(handle, t?HIPBLAS_OP_T:HIPBLAS_OP_N,m,n,alpha,A,lda,x,cx,beta,y,incy);
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("TIMES MM not working");
    }

    hipblasDestroy(handle);

    return;
}

void cuda_dgemv(bool t,int m, int n,const double * alpha,const double * A, int lda, const double * x, int cx, const double * beta,double * y,int incy)
{
    double * device_x; //vector
    double * device_A; //matrix
    double * device_y; //vector 

    hipMalloc(&device_x,(t?m:n)*sizeof(double));
    hipMalloc(&device_A,m*n*sizeof(double));
    hipMalloc(&device_y,(t?n:m)*sizeof(double));

    hipMemcpy(device_x,x,(t?m:n)*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_A,A,m* n * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_y,y,(t?n:m)*sizeof(double),hipMemcpyHostToDevice);

    cuda_gemv_helper(t,m,n,alpha,device_A,lda,device_x,cx,beta,device_y,incy);

    hipMemcpy(y,device_y,(t?n:m)*sizeof(double),hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(device_x);
    hipFree(device_A);
    hipFree(device_y);

    return;
}

void Execute_Helper_Kernel(int){
    //create cublas Handle here
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return;
    }else{
        printf("CUBLAS initialized succesfully");
    }
    //create and call the schedhuler here

    //return the result back into the parameter
    hipDeviceSynchronize();
} 
