#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "EXECUTE_HELPER.h"

__device__ int getGlobalIdx_1D_2D(){
    return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void cuda_Hello_World(){
    int tid = getGlobalIdx_1D_2D();
    printf("Hello World from thread ID:%d\n",tid);
}
//op_nop - no operator
//op_mat_inv -  matrix inversion
__device__ void op_mat_inv(){

}
//op_mat_mul - matrix multiplication
__device__ void op_mat_mul(){

}
//op mat add - matrix addition
__device__ void op_mat_add(){

}
//op_vec_mul - multiply two vectors
__device__ void op_vec_mul(){

}
//op_vec_add - add two vectors
__device__ void op_vec_add(){

}

void Execute_Helper_Kernel(int){
    //create cublas Handle here
    hipblasStatus_t status;
    hipblasHandle_t handle;
    status = hipblasCreate(&handle);
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return;
    }else{
        printf("CUBLAS initialized succesfully");
    }
    //create and call the schedhuler here

    //return the result back into the parameter
    hipDeviceSynchronize();
} 
